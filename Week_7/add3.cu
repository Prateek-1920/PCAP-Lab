#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void generateOutputStringKernel(const char* d_input, char* d_output, int input_length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < input_length) {
        int repeat_count = idx + 1;

        int start_pos = 0;
        for (int i = 0; i < idx; i++) {
            start_pos += (i + 1); 
        }

        for (int j = 0; j < repeat_count; j++) {
            d_output[start_pos + j] = d_input[idx];
        }
    }
}

int main() {
    char input[100];

    printf("Enter a string: ");
    fgets(input, sizeof(input), stdin);
    
    size_t len = strlen(input);
    if (len > 0 && input[len - 1] == '\n') {
        input[len - 1] = '\0';
    }

    int input_length = strlen(input);
    int output_length = (input_length * (input_length + 1)) / 2;

    char *d_input, *d_output;
    hipMalloc((void**)&d_input, input_length * sizeof(char));
    hipMalloc((void**)&d_output, (output_length + 1) * sizeof(char));

    hipMemcpy(d_input, input, input_length * sizeof(char), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (input_length + threadsPerBlock - 1) / threadsPerBlock;
    generateOutputStringKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, input_length);

    char* output = (char*)malloc((output_length + 1) * sizeof(char));  

    hipMemcpy(output, d_output, output_length * sizeof(char), hipMemcpyDeviceToHost);
    output[output_length] = '\0'; 

    printf("Output: %s\n", output);

    hipFree(d_input);
    hipFree(d_output);
    
    free(output);

    return 0;
}