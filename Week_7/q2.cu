#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void copyStringKernel(const char* S, char* RS, int lenS) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < lenS) {
        int numCharsToCopy = lenS - idx;

        int startPos = (lenS * (lenS + 1)) / 2 - (numCharsToCopy * (numCharsToCopy + 1)) / 2;

        for (int i = 0; i < numCharsToCopy; i++) {
            RS[startPos + i] = S[i];
        }
    }
}

int main() {
    const char* S = "PCAP";
    int lenS = strlen(S);
    int lenRS = (lenS * (lenS + 1)) / 2; 

    char* h_RS = (char*)malloc((lenRS + 1) * sizeof(char)); 
    h_RS[lenRS] = '\0';

    char *d_S, *d_RS;
    hipMalloc((void**)&d_S, lenS * sizeof(char));
    hipMalloc((void**)&d_RS, (lenRS + 1) * sizeof(char)); 

    hipMemcpy(d_S, S, lenS * sizeof(char), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (lenS + threadsPerBlock - 1) / threadsPerBlock;
    copyStringKernel<<<blocksPerGrid, threadsPerBlock>>>(d_S, d_RS, lenS);

    hipMemcpy(h_RS, d_RS, (lenRS + 1) * sizeof(char), hipMemcpyDeviceToHost);

    printf("Input S: %s\n", S);
    printf("Output RS: %s\n", h_RS);

    hipFree(d_S);
    hipFree(d_RS);
    
    free(h_RS);

    return 0;
}