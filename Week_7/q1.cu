#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ int compareStrings(const char* str1, const char* str2, int len) {
    for (int i = 0; i < len; i++) {
        if (str1[i] != str2[i]) {
            return 0; 
        }
    }
    return 1; 
}

__global__ void countWordOccurrences(char **d_words, int numWords, char *d_target, int targetLen, int *d_count) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < numWords) {
        if (compareStrings(d_words[idx], d_target, targetLen)) {
            atomicAdd(d_count, 1); 
        }
    }
}

int main() {
    char sequence[1024];
    printf("Enter a sequence of words (separated by spaces): ");
    fgets(sequence, sizeof(sequence), stdin);

    char targetWord[100];
    printf("Enter the word to count: ");
    fgets(targetWord, sizeof(targetWord), stdin);

    sequence[strcspn(sequence, "\n")] = '\0';
    targetWord[strcspn(targetWord, "\n")] = '\0';

    char *words[100];
    int numWords = 0;

    char *word = strtok(sequence, " ");
    while (word != NULL) {
        words[numWords++] = word;
        word = strtok(NULL, " ");
    }

    char **d_words;
    char *d_target;
    int *d_count;
    int targetLen = strlen(targetWord);

    hipMalloc((void**)&d_words, numWords * sizeof(char*));
    hipMalloc((void**)&d_target, targetLen * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_target, targetWord, targetLen * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(d_count, 0, sizeof(int));

    char **d_words_data;
    hipMalloc((void**)&d_words_data, numWords * sizeof(char*));

    for (int i = 0; i < numWords; i++) {
        char *d_word;
        hipMalloc((void**)&d_word, strlen(words[i]) + 1);
        hipMemcpy(d_word, words[i], strlen(words[i]) + 1, hipMemcpyHostToDevice);
        hipMemcpy(&d_words_data[i], &d_word, sizeof(char*), hipMemcpyHostToDevice);
    }
    
    hipMemcpy(d_words, d_words_data, numWords * sizeof(char*), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (numWords + blockSize - 1) / blockSize;

    countWordOccurrences<<<numBlocks, blockSize>>>(d_words, numWords, d_target, targetLen, d_count);

    int h_count = 0;
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the sequence.\n", targetWord, h_count);

    hipFree(d_words);
    hipFree(d_target);
    hipFree(d_count);
    hipFree(d_words_data);

    return 0;
}
