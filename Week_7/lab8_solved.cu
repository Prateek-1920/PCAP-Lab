#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void transposeKernel(int *d_A, int *d_AT, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        d_AT[col * m + row] = d_A[row * n + col];
    }
}

void printMatrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int m, n;
    printf("Enter number of rows (m): ");
    scanf("%d", &m);
    printf("Enter number of columns (n): ");
    scanf("%d", &n);

    int *h_A = (int *)malloc(m * n * sizeof(int));
    int *h_AT = (int *)malloc(n * m * sizeof(int));  

    printf("Enter the elements of the matrix A (%dx%d):\n", m, n);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("Element A[%d][%d]: ", i, j);
            scanf("%d", &h_A[i * n + j]);
        }
    }

    printf("\nOriginal Matrix A:\n");
    printMatrix(h_A, m, n);

    int *d_A, *d_AT;
    hipMalloc((void **)&d_A, m * n * sizeof(int));
    hipMalloc((void **)&d_AT, n * m * sizeof(int));

    hipMemcpy(d_A, h_A, m * n * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16); 
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y); 

    transposeKernel<<<gridDim, blockDim>>>(d_A, d_AT, m, n);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(h_AT, d_AT, n * m * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nTransposed Matrix AT:\n");
    printMatrix(h_AT, n, m);

    free(h_A);
    free(h_AT);
    hipFree(d_A);
    hipFree(d_AT);

    return 0;
}
