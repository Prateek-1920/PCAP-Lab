#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void countOccurrences(char *str, int *count, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < len) {
        if (str[idx] == 'a' ) {
            atomicAdd(count, 1);  
        }
    }
}

int main() {
    char str[100];
    printf("Enter string : ");
    fflush(stdout);
    scanf("%s",str);

    int len = strlen(str);

    char *d_str;
    int *d_count;
    int h_count = 0;

    hipMalloc((void**)&d_str, len * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_str, str, len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &h_count, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;  
    int numBlocks = (len + blockSize - 1) / blockSize;  

    countOccurrences<<<numBlocks, blockSize>>>(d_str, d_count, len);

    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("Number of occurrences of 'a': %d\n", h_count);

    hipFree(d_str);
    hipFree(d_count);

    return 0;
}
