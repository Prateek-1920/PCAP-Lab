#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void repeatStringKernel(char *d_Sin, char *d_Sout, int sin_len, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < sin_len * N) {
        int src_idx = idx % sin_len;  
        d_Sout[idx] = d_Sin[src_idx];
    }
}

void printString(char *str) {
    printf("%s", str);
}

int main() {
    char Sin[1000]; 
    int N;

    printf("Enter the string Sin: ");
    fgets(Sin, sizeof(Sin), stdin);

    Sin[strcspn(Sin, "\n")] = 0;

    printf("Enter the integer N: ");
    scanf("%d", &N);

    int sin_len = strlen(Sin);

    int sout_len = sin_len * N;
    char *d_Sin, *d_Sout;
    hipMalloc((void**)&d_Sin, sin_len * sizeof(char));
    hipMalloc((void**)&d_Sout, sout_len * sizeof(char));

    hipMemcpy(d_Sin, Sin, sin_len * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (sout_len + blockSize - 1) / blockSize;

    repeatStringKernel<<<numBlocks, blockSize>>>(d_Sin, d_Sout, sin_len, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    char Sout[1000]; 
    hipMemcpy(Sout, d_Sout, sout_len * sizeof(char), hipMemcpyDeviceToHost);

    Sout[sout_len] = '\0';

    printf("\nOutput string Sout: ");
    printString(Sout);
    printf("\n");

    hipFree(d_Sin);
    hipFree(d_Sout);

    return 0;
}
