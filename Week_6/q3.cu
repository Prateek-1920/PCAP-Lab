#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void odd_phase(int* arr, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx % 2 == 1 && idx < n - 1) {
        if (arr[idx] > arr[idx + 1]) {
            int temp = arr[idx];
            arr[idx] = arr[idx + 1];
            arr[idx + 1] = temp;
        }
    }
}

__global__ void even_phase(int* arr, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx % 2 == 0 && idx < n - 1) {
        if (arr[idx] > arr[idx + 1]) {
            int temp = arr[idx];
            arr[idx] = arr[idx + 1];
            arr[idx + 1] = temp;
        }
    }
}

void odd_even_transposition_sort(int* arr, int n) {
    int *d_arr;
    hipMalloc((void**)&d_arr, n * sizeof(int));

    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256; 
    int numBlocks = (n + blockSize - 1) / blockSize; 

    for (int phase = 0; phase < n; ++phase) {
        if (phase % 2 == 0) {
            even_phase<<<numBlocks, blockSize>>>(d_arr, n);
        } else {
            odd_phase<<<numBlocks, blockSize>>>(d_arr, n);
        }

        hipDeviceSynchronize();
    }

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_arr);
}

int main() {
    int n;

    printf("Enter the size of the array: ");
    scanf("%d", &n);

    int* arr = (int*)malloc(n * sizeof(int));

    printf("Enter %d elements for the array: ", n);
    for (int i = 0; i < n; i++) {
        scanf("%d", &arr[i]);
    }

    printf("Original Array:\n");
    for (int i = 0; i < n; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    odd_even_transposition_sort(arr, n);

    printf("Sorted Array:\n");
    for (int i = 0; i < n; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    free(arr);
    return 0;
}
