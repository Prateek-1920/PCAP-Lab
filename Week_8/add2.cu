#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int factorial(int n) {
    int result = 1;
    for (int i = 1; i <= n; i++) {
        result *= i;
    }
    return result;
}

__device__ int sum_of_digits(int num) {
    int sum = 0;
    while (num > 0) {
        sum += num % 10;
        num /= 10;
    }
    return sum;
}

__global__ void processMatrix(int *A, int *B, int n) {
    int row = threadIdx.x;
    int col = threadIdx.y;

    if (row < n && col < n) {
        if (row == col) {
            B[row * n + col] = 0;
        } else if (row < col) {
            B[row * n + col] = factorial(A[row * n + col]);
        } else {
            B[row * n + col] = sum_of_digits(A[row * n + col]);
        }
    }
}

int main() {
    int n;

    printf("Enter the size of the matrix (n x n): ");
    scanf("%d", &n);

    int A[n][n], B[n][n];

    printf("Enter elements of matrix A (%d x %d):\n", n, n);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("A[%d][%d]: ", i, j);
            scanf("%d", &A[i][j]);
        }
    }

    int *d_A, *d_B;
    size_t size_A = n * n * sizeof(int);
    size_t size_B = n * n * sizeof(int);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(n, n);
    processMatrix<<<1, threadsPerBlock>>>(d_A, d_B, n);

    hipMemcpy(B, d_B, size_B, hipMemcpyDeviceToHost);

    printf("Resultant Matrix B:\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
