#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void modifyMatrix(int *A, int *B, int m, int n) {
    int row = threadIdx.x;
    int col = threadIdx.y;

    if (row < m && col < n) {
        if (A[row * n + col] % 2 == 0) {
            int rowSum = 0;
            for (int j = 0; j < n; j++) {
                rowSum += A[row * n + j];
            }
            B[row * n + col] = rowSum;
        } else {
            int colSum = 0;
            for (int i = 0; i < m; i++) {
                colSum += A[i * n + col];
            }
            B[row * n + col] = colSum;
        }
    }
}

int main() {
    int m, n;

    printf("Enter the number of rows (m): ");
    scanf("%d", &m);
    printf("Enter the number of columns (n): ");
    scanf("%d", &n);

    int A[m][n], B[m][n];

    printf("Enter elements of matrix A (%d x %d):\n", m, n);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("A[%d][%d]: ", i, j);
            scanf("%d", &A[i][j]);
        }
    }

    int *d_A, *d_B;
    size_t size_A = m * n * sizeof(int);
    size_t size_B = m * n * sizeof(int);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(m, n);
    modifyMatrix<<<1, threadsPerBlock>>>(d_A, d_B, m, n);

    hipMemcpy(B, d_B, size_B, hipMemcpyDeviceToHost);

    printf("Resultant Matrix B:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
