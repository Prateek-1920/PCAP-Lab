#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void mulRows(int *A, int *B, int *C, int m, int n, int p) {
    int row = blockIdx.x;

    if (row < m) {
        for (int col = 0; col < p; ++col) {
            C[row * p + col] = 0;
            for (int k = 0; k < n; ++k) {
                C[row * p + col] += A[row * n + k] * B[k * p + col];
            }
        }
    }
}

__global__ void mulColumns(int *A, int *B, int *C, int m, int n, int p) {
    int col = blockIdx.x;

    if (col < p) {
        for (int row = 0; row < m; ++row) {
            C[row * p + col] = 0;
            for (int k = 0; k < n; ++k) {
                C[row * p + col] += A[row * n + k] * B[k * p + col];
            }
        }
    }
}

__global__ void mulElements(int *A, int *B, int *C, int m, int n, int p) {
    int row = threadIdx.x;
    int col = threadIdx.y;

    if (row < m && col < p) {
        C[row * p + col] = 0;
        for (int k = 0; k < n; ++k) {
            C[row * p + col] += A[row * n + k] * B[k * p + col];
        }
    }
}

int main() {
    int m, n, p;

    printf("Enter the number of rows for Matrix A (m): ");
    scanf("%d", &m);
    printf("Enter the number of columns for Matrix A (n): ");
    scanf("%d", &n);
    printf("Enter the number of columns for Matrix B (p): ");
    scanf("%d", &p);

    int A[m][n], B[n][p], C[m][p];

    printf("Enter elements of matrix A (%d x %d):\n", m, n);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("A[%d][%d]: ", i, j);
            scanf("%d", &A[i][j]);
        }
    }

    printf("Enter elements of matrix B (%d x %d):\n", n, p);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            printf("B[%d][%d]: ", i, j);
            scanf("%d", &B[i][j]);
        }
    }

    int *d_A, *d_B, *d_C;
    size_t size_A = m * n * sizeof(int);
    size_t size_B = n * p * sizeof(int);
    size_t size_C = m * p * sizeof(int);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    mulRows<<<m, 1>>>(d_A, d_B, d_C, m, n, p);

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    printf("Resultant Matrix C (Row-wise computation):\n");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < p; ++j) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    mulColumns<<<p, 1>>>(d_A, d_B, d_C, m, n, p);

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    printf("Resultant Matrix C (Column-wise computation):\n");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < p; ++j) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    dim3 threadsPerBlock(m, p);
    mulElements<<<1, threadsPerBlock>>>(d_A, d_B, d_C, m, n, p);

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    printf("Resultant Matrix C (Element-wise computation):\n");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < p; ++j) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
