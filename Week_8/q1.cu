#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addRows(int *A, int *B, int *C, int n) {
    int row = blockIdx.x;

    if (row < n) {
        for (int col = 0; col < n; ++col) {
            C[row * n + col] = A[row * n + col] + B[row * n + col];
        }
    }
}

__global__ void addColumns(int *A, int *B, int *C, int n) {
    int col = blockIdx.x;

    if (col < n) {
        for (int row = 0; row < n; ++row) {
            C[row * n + col] = A[row * n + col] + B[row * n + col];
        }
    }
}

__global__ void addElements(int *A, int *B, int *C, int n) {
    int row = threadIdx.x;
    int col = threadIdx.y;

    if (row < n && col < n) {
        C[row * n + col] = A[row * n + col] + B[row * n + col];
    }
}

int main() {
    int n;

    printf("Enter the size of the matrix (n x n): ");
    scanf("%d", &n);

    int A[n][n], B[n][n], C[n][n];
    
    printf("Enter elements of matrix A (%d x %d):\n", n, n);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("A[%d][%d]: ", i, j);
            scanf("%d", &A[i][j]);
        }
    }

    printf("Enter elements of matrix B (%d x %d):\n", n, n);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("B[%d][%d]: ", i, j);
            scanf("%d", &B[i][j]);
        }
    }

    int *d_A, *d_B, *d_C;
    size_t size = n * n * sizeof(int);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    addRows<<<n, 1>>>(d_A, d_B, d_C, n);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix C (Row-wise computation):\n");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    addColumns<<<n, 1>>>(d_A, d_B, d_C, n);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix C (Column-wise computation):\n");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    dim3 threadsPerBlock(n, n);
    addElements<<<1, threadsPerBlock>>>(d_A, d_B, d_C, n);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix C (Element-wise computation):\n");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
