#include <stdio.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256 // Number of threads per block

__global__ void vectorAddKernel(float *A, float *B, float *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
    if (i < N)
    {                       // Ensure the thread is within bounds of the vector length
        C[i] = A[i] + B[i]; // Perform vector addition
        printf("Thread %d: A[%d] = %f, B[%d] = %f, C[%d] = %f\n", threadIdx.x, i, A[i], i, B[i], i, C[i]);
    }
}

void vectorAdd(float *A, float *B, float *C, int N)
{
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));

    // Copy vectors from host to device
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Calculate the number of blocks needed (round up if necessary)
    int numBlocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel with numBlocks and THREADS_PER_BLOCK threads per block
    vectorAddKernel<<<numBlocks, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main()
{
    int N = 5; // Length of the vectors (you can change this value)

    float *A = (float *)malloc(N * sizeof(float));
    float *B = (float *)malloc(N * sizeof(float));
    float *C = (float *)malloc(N * sizeof(float));

    // Initialize vectors A and B
    for (int i = 0; i < N; i++)
    {
        A[i] = i * 1.0f; // Fill A with values 0, 1, 2, 3, ...
        B[i] = i * 2.0f; // Fill B with values 0, 2, 4, 6, ...
    }

    // Add vectors
    vectorAdd(A, B, C, N);

    // Print the result of the addition
    printf("\nFinal result:\n");
    for (int i = 0; i < N; i++)
    {
        printf("%f + %f = %f\n", A[i], B[i], C[i]);
    }

    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
