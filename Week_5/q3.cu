#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h> // For sinf() function

// Kernel to compute sine of each angle in the input array
__global__ void computeSine(float *angles, float *sineValues, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index

    // Process only valid indices within the array
    if (idx < N)
    {
        sineValues[idx] = sinf(angles[idx]); // Calculate sine for each angle
        printf("Thread %d: Sine of angle %.3f radians = %.3f\n", threadIdx.x, angles[idx], sineValues[idx]);
    }
}

void processAngles(float *angles, float *sineValues, int N)
{
    float *d_angles, *d_sineValues;

    // Allocate device memory for input and output arrays
    hipMalloc((void **)&d_angles, N * sizeof(float));
    hipMalloc((void **)&d_sineValues, N * sizeof(float));

    // Copy input array from host to device
    hipMemcpy(d_angles, angles, N * sizeof(float), hipMemcpyHostToDevice);

    // Calculate number of blocks and threads
    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel to compute sine values
    computeSine<<<numBlocks, threadsPerBlock>>>(d_angles, d_sineValues, N);

    // Copy result from device to host
    hipMemcpy(sineValues, d_sineValues, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_angles);
    hipFree(d_sineValues);
}

int main()
{
    int N = 5; // Length of the array (you can change this to any value)

    float *angles = (float *)malloc(N * sizeof(float));
    float *sineValues = (float *)malloc(N * sizeof(float));

    // Initialize angles array (example: 0, PI/6, PI/4, PI/2, ...)
    for (int i = 0; i < N; i++)
    {
        angles[i] = i * 3.14159265358979f / 6.0f; // Example angles in radians (multiples of pi/6)
    }

    // Process angles and compute sine values
    processAngles(angles, sineValues, N);

    // Print results
    printf("Angle (radians)     Sine of Angle\n");
    for (int i = 0; i < N; i++)
    {
        printf("%.3f               %.3f\n", angles[i], sineValues[i]);
    }

    // Free host memory
    free(angles);
    free(sineValues);

    return 0;
}
