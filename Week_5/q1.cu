#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 5 // Length of the vectors

__global__ void vectorAddKernel(float *A, float *B, float *C)
{
    int i = threadIdx.x; // Each thread handles one element
    if (i < N)
    {
        C[i] = A[i] + B[i];
        printf("Thread %d: A[%d] = %f, B[%d] = %f, C[%d] = %f\n", i, i, A[i], i, B[i], i, C[i]);
    }
}

void vectorAdd(float *A, float *B, float *C)
{
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));

    // Copy vectors from host to device
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with 1 block of N threads
    vectorAddKernel<<<1, N>>>(d_A, d_B, d_C);

    // Copy result from device to host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main()
{
    float A[N], B[N], C[N];

    // Initialize vectors A and B
    for (int i = 0; i < N; i++)
    {
        A[i] = i;
        B[i] = i * 2;
    }

    // Add vectors
    vectorAdd(A, B, C);

    // Print result
    printf("\nFinal result:\n");
    for (int i = 0; i < N; i++) {
        printf("%f + %f = %f\n", A[i], B[i], C[i]);
    }

    return 0;
}