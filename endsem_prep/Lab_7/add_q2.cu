#include<stdio.h>
#include<hip/hip_runtime.h>
#include<string.h>

__global__ void multiply(char *str, char *res, int len, int k){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < len){
        for(int i=0;i<k;i++){
            res[idx+(i*len)]=str[idx];
        }
    }
}

int main(){
    char s[100];
    int n;
    printf("Enter string: ");
    scanf("%s",s);

    printf("Enter n: ");
    scanf("%d",&n);

    int len = strlen(s);
    s[len] = '\0';

    char *d_s, *res;
    hipMalloc((void**)&d_s,len*sizeof(char));
    hipMalloc((void**)&res,len*n*sizeof(char));

    hipMemcpy(d_s,s,len*sizeof(char),hipMemcpyHostToDevice);

    int THREADS = 256;
    int numblocks = (len + THREADS -1 )/ THREADS;

    multiply<<<numblocks,THREADS>>>(d_s,res,len,n);

    char ans[100];
    hipMemcpy(ans,res,len*n*sizeof(char),hipMemcpyDeviceToHost);
    ans[len*n] = '\0';

    printf("Original string : %s \nResult string : %s \n",s,ans);

    hipFree(d_s);
    hipFree(res);

    return 0;

}


