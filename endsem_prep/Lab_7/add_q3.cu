#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>

__global__ void mult(char *s, char *res, int len){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < len){
        int repeatcount = idx+1;
        int start_pos = 0;
        for (int i = 0; i < idx; i++) {
            start_pos += (i + 1); 
        }

        for (int j = 0; j < repeatcount; j++) {
            res[start_pos + j] = s[idx];
        }
    }
}

int main(){
    char s[100];
    printf("Enter string: ");
    scanf("%s",s);
    
    int len = strlen(s);
    s[len] = '\0';

    int reslen = (len * (len+1))/2;
    char *d_s, *res;
    hipMalloc((void**)&d_s,len*sizeof(char));
    hipMalloc((void**)&res,reslen*sizeof(char));

    hipMemcpy(d_s,s,len*sizeof(char),hipMemcpyHostToDevice);

    int THREADS = 256;
    int numblocks = (len + THREADS -1 )/ THREADS;

    mult<<<numblocks,THREADS>>>(d_s,res,len);

    char ans[100];
    hipMemcpy(ans,res,reslen*sizeof(char),hipMemcpyDeviceToHost);
    ans[reslen] = '\0';

    printf("Original string : %s \nResult string : %s \n",s,ans);

    hipFree(d_s);
    hipFree(res);
}
