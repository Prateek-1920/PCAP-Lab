#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>


__global__ void stringcat(char *s, char *res, int len){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx<len){
        int copylen = len-idx;
        int startpos = (len * (len + 1)) / 2 - (copylen * (copylen+ 1)) / 2;

        for(int i=0;i<copylen;i++){
            res[i+startpos] = s[i];
        }
    }
}


int main(){
    char s[100];
    printf("Enter string: ");
    scanf("%s",s);
    
    int len = strlen(s);
    s[len]='\0';
    int reslen = (len * (len+1))/2;

    char *d_s, *res;
    hipMalloc((void**)&d_s,len*sizeof(char));
    hipMalloc((void**)&res,reslen*sizeof(char));

    hipMemcpy(d_s,s,len*sizeof(char),hipMemcpyHostToDevice);

    int THREADS_PER_BLOCK = 256;
    int numblocks = (len + THREADS_PER_BLOCK -1) /THREADS_PER_BLOCK;

    stringcat<<<numblocks,THREADS_PER_BLOCK>>>(d_s,res,len);

    char ans[100];
    hipMemcpy(ans,res,reslen*sizeof(char),hipMemcpyDeviceToHost);
    ans[reslen] = '\0';

    printf("Resultant string : %s\n",ans);

    hipFree(d_s);
    hipFree(res);

    return 0;

}