#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void onescomp(int *a, int* b, int n){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n){
        b[idx] = ~a[idx] & 0b11111; // Mask to keep only the lower 5 bits
    }
}

int main(){
    int bin[5] = {0b11011,0b10101,0b11100,0b00101,0b00100}; //binary literals
    int n = sizeof(bin) / sizeof(bin[0]);

    int *d_a, *d_b;
    hipMalloc((void**)&d_a,n*sizeof(int));
    hipMalloc((void**)&d_b,n*sizeof(int));

    hipMemcpy(d_a,bin,n*sizeof(int),hipMemcpyHostToDevice);

    int THREADS_PER_BLOCK = 256;
    int numblocks = n+THREADS_PER_BLOCK-1/THREADS_PER_BLOCK;

    onescomp<<<numblocks,THREADS_PER_BLOCK>>>(d_a,d_b,n);

    int res[5];
    hipMemcpy(res,d_b,n*sizeof(int),hipMemcpyDeviceToHost);

    printf("Original  |  Ones complement : \n");
    for(int i=0;i<n;i++){
        printf("%5d  %5d\n",bin[i],res[i]);
    }

    hipFree(d_a);
    hipFree(d_b);

    return 0;

}