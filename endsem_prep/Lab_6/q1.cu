#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void conv(float *org, float *mask, float *res, int width, int mask_width) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    int output_width = width - mask_width + 1;

    if (idx < output_width) {
        float ans = 0.0;
        for (int i = 0; i < mask_width; i++) {
            ans += org[idx + i] * mask[i];
        }
        res[idx] = ans;
    }
}

int main() {
    float org[5] = {1, 2, 3, 4, 5};
    float mask[3] = {3, 3, 3};

    int width = sizeof(org) / sizeof(org[0]);
    int mask_width = sizeof(mask) / sizeof(mask[0]);
    int output_width = width - mask_width + 1;

    float *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, width * sizeof(float));
    hipMalloc((void**)&d_b, mask_width * sizeof(float));
    hipMalloc((void**)&d_c, output_width * sizeof(float));

    hipMemcpy(d_a, org, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, mask, mask_width * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (output_width + threadsPerBlock - 1) / threadsPerBlock;
    conv<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, width, mask_width);

    float res[output_width];
    hipMemcpy(res, d_c, output_width * sizeof(float), hipMemcpyDeviceToHost);

    printf("Original Array: ");
    for (int i = 0; i < width; i++) {
        printf("%f  ", org[i]);
    }

    printf("\nMask: ");
    for (int i = 0; i < mask_width; i++) {
        printf("%f  ", mask[i]);
    }

    printf("\nResultant Array: ");
    for (int i = 0; i < output_width; i++) {
        printf("%f  ", res[i]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
