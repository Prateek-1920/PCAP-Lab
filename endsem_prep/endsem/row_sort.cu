#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void sort(int *A, int rows, int cols){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    if(row<rows){
        for(int i=0;i<cols;i++){
            for(int j=0;j<cols-1-i;j++){
            int idx = row * cols + j;
            if(A[idx] > A[idx+1]){
                int temp = A[idx];
                A[idx] = A[idx+1];
                A[idx+1] = temp;
            }
            }
        }
    }
}

int main(){

    hipEvent_t start,stop;
    float ms = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int m,n;
    printf("Enter m and n: ");
    scanf("%d%d",&m,&n);

    int A[m][n] , B[m][n];
    printf("Enter elements: ");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            scanf("%d",&A[i][j]);
        }
    }

    int *d_a;
    hipMalloc((void**)&d_a,m*n*sizeof(int));
    hipMemcpy(d_a,A,n*m*sizeof(int),hipMemcpyHostToDevice);

    dim3 blockspergrid(3,3);
    dim3 threadsperblock(16,16);

    hipEventRecord(start);
    sort<<<blockspergrid,threadsperblock>>>(d_a,m,n);
    hipDeviceSynchronize();
    hipEventRecord(stop);


    hipMemcpy(B,d_a,m*n*sizeof(int),hipMemcpyDeviceToHost);

    printf("\nMatrix before row sorting\n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d  ",A[i][j]);
        }
        printf("\n");
    }

    printf("\nMatrix after row sorting\n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d  ",B[i][j]);
        }
        printf("\n");
    }

    hipEventElapsedTime(&ms,start,stop);
    printf("Time taken = %f",ms);

    hipFree(d_a);
    return 0;
}