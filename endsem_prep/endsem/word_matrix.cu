#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_WORD_LENGTH 20

__device__ int d_found = -1;

__global__ void findword(char *A, char *B, int rows, int cols, int wordLength) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int idx = row * cols + col;
        bool match = true; 
        
        // Compare each character of A[idx] with B
        for (int i = 0; i < wordLength; i++) {
            if (A[idx * MAX_WORD_LENGTH + i] != B[i]) {
                match = false;
                break;
            }
        }

        // Check if both words end at same length
        if (match && A[idx * MAX_WORD_LENGTH + wordLength] == '\0') {
            atomicExch(&d_found, idx);
        }
    }
}

int main() {
    int m, n;
    printf("Enter m and n: ");
    scanf("%d%d", &m, &n);

    char A[m][n][MAX_WORD_LENGTH];
    printf("Enter words:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            scanf("%s", A[i][j]);
        }
    }

    char B[MAX_WORD_LENGTH];
    printf("Enter target word: ");
    scanf("%s", B);  
    int len = strlen(B);

    // Flatten A
    char flatA[m * n * MAX_WORD_LENGTH];
    for (int i = 0; i < m * n; ++i) {
        strncpy(&flatA[i * MAX_WORD_LENGTH], A[i / n][i % n], MAX_WORD_LENGTH);
    }

    // Allocate device memory
    char *d_a, *d_b;
    hipMalloc((void**)&d_a, m * n * MAX_WORD_LENGTH * sizeof(char));
    hipMalloc((void**)&d_b, MAX_WORD_LENGTH * sizeof(char));

    hipMemcpy(d_a, flatA, m * n * MAX_WORD_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, MAX_WORD_LENGTH * sizeof(char), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(4,4);
    findword<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, m, n, len);

    // Copy back result
    int h_found = -1;
    hipMemcpyFromSymbol(&h_found, HIP_SYMBOL(d_found), sizeof(int), 0, hipMemcpyDeviceToHost);

    if (h_found != -1) {
        printf("Word '%s' found at index %d (row = %d, col = %d)\n", B, h_found, h_found / n, h_found % n);
    } else {
        printf("Word '%s' not found\n", B);
    }

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
