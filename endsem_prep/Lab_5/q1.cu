#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void vectoradd(int *a, int *b, int *c,int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<n){
        c[idx] = a[idx] + b[idx];
    }
}

int main(){
    int n;

    printf("Enter N: ");
    scanf("%d",&n);

    int A[n],B[n],C[n];

    printf("Enter elements of A: ");
    for(int i=0;i<n;i++){
        scanf("%d",&A[i]);
    }

    printf("Enter elements of B: ");
    for(int i=0;i<n;i++){
        scanf("%d",&B[i]);
    }

    int *d_a,*d_b,*d_c;
    hipMalloc((void**)&d_a,n*sizeof(int));
    hipMalloc((void**)&d_b,n*sizeof(int));
    hipMalloc((void**)&d_c,n*sizeof(int));

    hipMemcpy(d_a,A,n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,B,n*sizeof(int),hipMemcpyHostToDevice);

    vectoradd<<<1,n>>>(d_a,d_b,d_c,n);

    hipMemcpy(C,d_c,n*sizeof(int),hipMemcpyDeviceToHost);

    printf("A array : ");
    for(int i=0;i<n;i++){
        printf("%d  ",A[i]);
    }

    printf("\nB array : ");
        for(int i=0;i<n;i++){
            printf("%d  ",B[i]);
        }

    printf("\nResultant array : ");
    for(int i=0;i<n;i++){
        printf("%d  ",C[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;

}