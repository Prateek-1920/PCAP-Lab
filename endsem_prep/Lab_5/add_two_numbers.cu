#include <stdio.h>
#include <hip/hip_runtime.h>  // ✅ CUDA runtime header

// CUDA kernel to add two numbers
__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int a = 10, b = 20, c;

    int *d_a, *d_b, *d_c;

    // Allocate memory on the GPU
    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));

    // Copy input values to the GPU
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel (1 block, 1 thread)
    add<<<1,1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum = %d\n", c);

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
