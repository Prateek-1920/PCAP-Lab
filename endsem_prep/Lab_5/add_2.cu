#include<stdio.h>
#include<hip/hip_runtime.h>

#define THREADS_PER_BLOCK 8

__global__ void oddeven(int *a, int n, int phase){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i = phase%2==0? 2*idx : 2*idx+1;

    if(i+1<n){
        if(a[i+1]<a[i]){
            int temp = a[i+1];
            a[i+1] = a[i];
            a[i] = temp;
        }
    }
}

int main(){
    int n;
    printf("Enter n: ");
    scanf("%d",&n);

    int A[n];
    printf("Enter elements: ");
    for(int i=0;i<n;i++){
        scanf("%d",&A[i]);
    }

    int *d_a;
    hipMalloc((void**)&d_a,n*sizeof(int));

    hipMemcpy(d_a,A,n*sizeof(int),hipMemcpyHostToDevice);

    int numblocks = n + THREADS_PER_BLOCK -1 /THREADS_PER_BLOCK;

    printf("Array before sorting : ");
    for(int i=0;i<n;i++){
        printf("%d  ",A[i]);
    }

    for(int phase=0;phase<n;phase++){
        oddeven<<<numblocks,THREADS_PER_BLOCK>>>(d_a,n,phase);
    }
    hipMemcpy(A,d_a,n*sizeof(int),hipMemcpyDeviceToHost);

    printf("\nArray after sorting : ");
    for(int i=0;i<n;i++){
        printf("%d  ",A[i]);
    }

    hipFree(d_a);
    return 0;


}