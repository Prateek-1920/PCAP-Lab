#include<stdio.h>
#include<hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256

__global__ void sort(int *matrix, int rows, int cols){
    int row = blockIdx.x;
    int idx = threadIdx.x;

    if(row<rows){
        for(int i=0;i<cols-1;i++){
            if(idx==i){
                int minidx = i;
                for(int j=i+1;j<cols;j++){
                    if(matrix[row*cols+j] < matrix[row*cols+minidx]){
                        minidx = j;
                    }
                }

                if(minidx!=i){
                    int temp = matrix[row*cols+i];
                    matrix[row*cols+i] = matrix[row*cols+minidx];
                    matrix[row*cols+minidx] = temp;
                }
            }
        }
    }
}

int main(){
    int m,n;
    printf("Enter m and n: ");
    scanf("%d%d",&m,&n);

    int A[m][n];

    printf("Enter matrix: \n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            scanf("%d",&A[i][j]);
        }
    }

    printf("Matrix before sorting\n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d  ",A[i][j]);
        }
        printf("\n");
    }


    int *d_a;
    size_t size = m*n*sizeof(int);
    hipMalloc((void**)&d_a,size);

    hipMemcpy(d_a,A,size,hipMemcpyHostToDevice);
    
    sort<<<m,n>>>(d_a,m,n);

    hipMemcpy(A,d_a,size,hipMemcpyDeviceToHost);

    printf("Matrix after sorting\n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d  ",A[i][j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    return 0;


}