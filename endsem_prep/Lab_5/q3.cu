#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

__global__ void sine(float *a,float *b, int n){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx<n){
        b[idx] = sinf(a[idx]);
    }
}

int main(){
    int n;
    printf("Enter n: ");
    scanf("%d",&n);
    float A[n];
    float B[n];

    printf("Enter values: ");
    for(int i=0;i<n;i++){
        scanf("%f",&A[i]);
    }

    float *d_a, *d_b;
    hipMalloc((void**)&d_a,n*sizeof(float));
    hipMalloc((void**)&d_b,n*sizeof(float));

    hipMemcpy(d_a,A,n*sizeof(float),hipMemcpyHostToDevice);

    sine<<<1,n>>>(d_a,d_b,n);

    hipMemcpy(B,d_b,n*sizeof(float),hipMemcpyDeviceToHost);

    printf("Sine values: ");
    for(int i=0;i<n;i++){
        printf("%f",B[i]);
    }

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}