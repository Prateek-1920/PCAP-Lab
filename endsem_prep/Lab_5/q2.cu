#include<stdio.h>
#include<hip/hip_runtime.h>

# define THREADS_PER_BLOCK 256

__global__ void vectoradd(int *a, int *b, int *c){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<THREADS_PER_BLOCK){
        c[idx] = a[idx] + b[idx];
    }
}

int main(){
    int n;
    printf("Enter N: ");
    scanf("%d",&n);

    int A[n], B[n], C[n];

    printf("Enter elements of A: ");
    for(int i=0;i<n;i++){
        scanf("%d",&A[i]);
    }

    printf("Enter elements of B: ");
    for(int i=0;i<n;i++){
        scanf("%d",&B[i]);
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a,n*sizeof(int));
    hipMalloc((void**)&d_b,n*sizeof(int));
    hipMalloc((void**)&d_c,n*sizeof(int));

    hipMemcpy(d_a,A,n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,B,n*sizeof(int),hipMemcpyHostToDevice);

    // numBlocks = N / THREADS_PER_BLOCK;
    // But if it's not evenly divisible, you'll need one extra block to cover the remaining threads.


    int numblocks = (n +THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    vectoradd<<<numblocks,THREADS_PER_BLOCK>>>(d_a,d_b,d_c);

    hipMemcpy(C,d_c,n*sizeof(int),hipMemcpyDeviceToHost);

    printf("A array : ");
    for(int i=0;i<n;i++){
        printf("%d  ",A[i]);
    }

    printf("\nB array : ");
        for(int i=0;i<n;i++){
            printf("%d  ",B[i]);
        }

    printf("\nResultant array : ");
    for(int i=0;i<n;i++){
        printf("%d  ",C[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
    
}