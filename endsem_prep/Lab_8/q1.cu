#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void addmatrix(int *A, int *B, int *C, int rows, int cols){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if(row < rows && col < cols){
        int idx = row*cols + col;
        C[idx] = A[idx] + B[idx];
    }
}

int main(){
    int m,n;
    printf("Enter rows and cols: ");
    scanf("%d%d",&m,&n);

    int A[m][n],B[m][n],C[m][n];
    printf("Enter elemenets 1: ");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            scanf("%d",&A[i][j]);
        }
    }
    printf("Enter elemenets 2: ");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            scanf("%d",&B[i][j]);
        }
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a,m*n*sizeof(int));
    hipMalloc((void**)&d_b,m*n*sizeof(int));
    hipMalloc((void**)&d_c,m*n*sizeof(int));
    hipMemcpy(d_a,A,m*n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,B,m*n*sizeof(int),hipMemcpyHostToDevice);

    dim3 blockspergrid(4,4);
    dim3 threadsperblock(16,16);
    addmatrix<<<blockspergrid,threadsperblock>>>(d_a,d_b,d_c,m,n);

    hipMemcpy(C,d_c,m*n*sizeof(int),hipMemcpyDeviceToHost);
    printf("Matrix A : \n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d  ",A[i][j]);
        }
        printf("\n");
    }

    printf("\nMatrix B : \n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d  ",B[i][j]);
        }
        printf("\n");
    }

    printf("\nMatrix C : \n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d  ",C[i][j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;

}