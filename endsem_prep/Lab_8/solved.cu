#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// #define N 4  // Rows
// #define M 3  // Columns

__global__ void transpose(int *input, int *output, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Row index
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Column index

    if (row < rows && col < cols) {
        int inputIdx = row * cols + col;              // Index in input matrix
        int outputIdx = col * rows + row;             // Index in transposed matrix
        output[outputIdx] = input[inputIdx];
    }
}

int main() {
    // int h_input[N][M] = {
    //     {1, 2, 3},
    //     {4, 5, 6},
    //     {7, 8, 9},
    //     {10,11,12}
    // };

    printf("Enter rows and cols: ");
    scanf("%d%d",&m,&n);
    int h_input[m][n];
    printf("Enter matrix elements: ");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            scanf("%d",&h_input[m][n]);
        }
    }

    int h_output[m][n];  // Transposed output

    int *d_input, *d_output;
    hipMalloc((void**)&d_input, m*n*sizeof(int));
    hipMalloc((void**)&d_output, m*n*sizeof(int));

    hipMemcpy(d_input, h_input, m*n*sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((m + threadsPerBlock.x - 1)/threadsPerBlock.x,
                   (n + threadsPerBlock.y - 1)/threadsPerBlock.y);

    transpose<<<numBlocks, threadsPerBlock>>>(d_input, d_output, n, m);

    hipMemcpy(h_output, d_output, n*m*sizeof(int), hipMemcpyDeviceToHost);

    printf("Original Matrix:\n");
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++)
            printf("%d ", h_input[i][j]);
        printf("\n");
    }

    printf("\nTransposed Matrix:\n");
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++)
            printf("%d ", h_output[i][j]);
        printf("\n");
    }

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
